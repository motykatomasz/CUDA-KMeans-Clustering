#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "Utils.hpp"
#include "CudaErrors.hpp"

void prepare(const std::shared_ptr<DataSet> &dataSet, const float* const hCentroids, float** dDataSet, unsigned int** dLabels, 
        float** dCentroids, bool** dUpdated, unsigned int k) {

    size_t numSamples = dataSet->samples.size();
    size_t sizeFeatureVector = dataSet->numFeatures * sizeof(float);

    float* tempDataSet, *tempCentroids;
    unsigned int* tempLabels;
    bool* tempUpdated;

    checkCudaErrors(hipMalloc(&tempDataSet, numSamples * sizeFeatureVector));
    checkCudaErrors(hipMalloc(&tempLabels, numSamples * sizeof(unsigned int)));
    checkCudaErrors(hipMalloc(&tempCentroids, k * sizeFeatureVector));
    checkCudaErrors(hipMalloc(&tempUpdated, sizeof(bool)));


    checkCudaErrors(hipMemcpy(tempDataSet, dataSet->samplesArray, numSamples * sizeFeatureVector, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(tempCentroids, hCentroids, k * sizeFeatureVector, hipMemcpyHostToDevice));

    *dDataSet = tempDataSet;
    *dLabels = tempLabels;
    *dCentroids = tempCentroids;
    *dUpdated = tempUpdated;
}


void clearPointers(float* d_dataset, unsigned int* d_labels, float* d_centroids, bool* d_updated, unsigned int* h_labels, float* h_centroids, unsigned int numSamples,
        unsigned int k, unsigned int numFeatures) {

    std::cout << h_labels << std::endl;

    checkCudaErrors(hipMemcpy(h_labels, d_labels, numSamples * sizeof(unsigned int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_centroids, d_centroids, k * numFeatures * sizeof(float), hipMemcpyDeviceToHost));

    // std::cout << "Centroids after copy (1):" << std::endl;
    // for (int i=0; i < k; ++i){
    //     std::cout << "Centroid " << i << ":";
    //     for (int j=0; j < numFeatures; ++j){
    //         std::cout << h_centroids[i*numFeatures + j] << " ";
    //     }
    //     std::cout << std::endl;
    // }


    // for (int i=0; i< numSamples; ++i){
    //     std::cout << h_labels[i] << " ";
    // }

    // std::cout << std::endl;

    checkCudaErrors(hipFree(d_dataset));
    checkCudaErrors(hipFree(d_labels));
    checkCudaErrors(hipFree(d_centroids));
    checkCudaErrors(hipFree(d_updated));
}